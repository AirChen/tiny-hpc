#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel()
{
    printf("hello world\n");
}

int main()
{
    kernel<<<10,10>>>();
    hipDeviceSynchronize();

    return 0;
}